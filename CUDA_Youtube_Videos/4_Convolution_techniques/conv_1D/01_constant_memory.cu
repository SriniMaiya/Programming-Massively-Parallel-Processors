
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <vector>
#include <algorithm>

#define MASK_SHAPE 7
__constant__ int mask[MASK_SHAPE];

__global__ void conv1D(const int *input, int *result, const int n_inp)
{
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    int mask_radius = MASK_SHAPE / 2;
    int mask_start = global_tid - mask_radius;

    for (int j = 0; j < MASK_SHAPE; j++)
    {
        if (((mask_start + j) >= 0) && ((mask_start + j) < n_inp))
        {
            sum += mask[j] * input[mask_start + j];
        }
    }

    result[global_tid] = sum;
}

void verify_results(const int *input, const int *mask, const int *result,
                    const int n_inp, const int n_mask)
{
    for (int i = 0; i < n_inp; i++)
    {
        int sum = 0;
        int mask_radius = n_mask / 2;
        int mask_start = i - mask_radius;

        for (int j = 0; j < n_mask; j++)
        {
            if (((mask_start + j) >= 0) && ((mask_start + j) < n_inp))
            {
                sum += mask[j] * input[mask_start + j];
            }
        }
        // printf("CPU: %d, GPU: %d \n", sum, result[i]);
        assert(sum == result[i]);
    }
}

int main()
{
    int n_inp = 2 << 20;
    int n_mask = 7;
    int bytes_inp = n_inp * sizeof(int);
    int bytes_mask = n_mask * sizeof(int);

    // Host data.
    std::vector<int> h_inp(n_inp), h_mask(n_mask), h_res(n_inp);

    std::generate(h_inp.begin(), h_inp.end(), []()
                  { return rand() % 10; });
    h_mask = {1, 1, 1, 2, 1, 1, 1};

    // Device data.
    int *d_inp, *d_mask, *d_res;

    hipMalloc(&d_inp, bytes_inp);
    hipMalloc(&d_res, bytes_inp);
    hipMalloc(&d_mask, bytes_mask);

    // Copy host -> device.
    hipMemcpy(d_inp, h_inp.data(), bytes_inp, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask.data(), bytes_mask, 0, hipMemcpyHostToDevice);

    int BLOCK_SIZE = 256;
    int GRID_SIZE = (n_inp + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Kernel run.
    conv1D<<<GRID_SIZE, BLOCK_SIZE>>>(d_inp, d_res, n_inp);

    hipDeviceSynchronize();

    // Copy device -> host.
    hipMemcpy(h_res.data(), d_res, bytes_inp, hipMemcpyDeviceToHost);

    verify_results(h_inp.data(), h_mask.data(), h_res.data(), n_inp, n_mask);

    hipFree(d_inp);
    hipFree(d_mask);
    hipFree(d_res);

    return 0;
}