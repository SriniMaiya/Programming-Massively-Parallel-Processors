
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int data_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (data_index < n)
    {
        C[data_index] = A[data_index] + B[data_index];
    }
}

void vecAdd(float *A, float *B, float *C, int n)
{
    float *A_d, *B_d, *C_d;
    int bytes = n * sizeof(float);

    hipMalloc(&A_d, bytes);
    hipMalloc(&B_d, bytes);
    hipMalloc(&C_d, bytes);

    hipMemcpy(A_d, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, bytes, hipMemcpyHostToDevice);

    int THREADBLOCK_SIZE = 256;
    int GRID_SIZE = (n + THREADBLOCK_SIZE - 1) / THREADBLOCK_SIZE;
    // int GRID_SIZE = (int)ceil((float)n / THREADBLOCK_SIZE);

    vecAddKernel<<<GRID_SIZE, THREADBLOCK_SIZE>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, bytes, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
}