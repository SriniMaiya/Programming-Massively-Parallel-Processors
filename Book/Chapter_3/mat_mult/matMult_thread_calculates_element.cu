
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
// #define STB_IMAGE_IMPLEMENTATION
// #include "stb_image.h"
// #define STB_IMAGE_WRITE_IMPLEMENTATION
// #include "stb_image_write.h"

void print_mat(float *, int, int);

__global__ void matMultKernel(float *M, float *N, float *P, int height_m, int k, int width_n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < height_m) && (col < width_n))
    {
        float sum = 0;
        for (int K = 0; K < k; K++)
        {
            sum += M[row * k + K] * N[K * width_n + col];
        }
        P[row * width_n + col] = sum;
    }
}

void init_mat(float *mat, int num_elements)
{
    for (int i = 0; i < num_elements; i++)
    {
        mat[i] = (float)rand() / (float)RAND_MAX;
    }
}

void check_results(float *M, float *N, float *P, int height_m, int k, int width_n)
{
    // printf("::: M :::\n");
    // print_mat(M, height_m, k);
    // printf("::: N :::\n");
    // print_mat(N, k, width_n);
    // printf("::: P :::\n");
    // print_mat(P, height_m, width_n);
    // printf("\n");
    for (int r_m = 0; r_m < height_m; r_m++)
    {
        for (int c_n = 0; c_n < width_n; c_n++)
        {
            float sum = 0;
            for (int K = 0; K < k; K++)
            {
                sum += M[r_m * k + K] * N[K * width_n + c_n];
            }
            // printf("%f == %f", sum, P[r_m * width_n + c_n]);
            // printf("%f", sum);
            // if (c_n == (width_n - 1))
            // {
            //     printf("\n");
            // }
            // else
            // {
            //     printf(" ");
            // }
            assert(abs(sum - P[r_m * width_n + c_n]) < 1e-4);
        }
    }
}

void print_mat(float *mat, int rows, int cols)
{
    for (int r = 0; r < rows; r++)
    {
        for (int c = 0; c < cols; c++)
        {
            printf("%f ", mat[r * cols + c]);
        }
        printf("\n");
    }
}

int main()
{
    float *M_d, *N_d, *P_d, *M_h, *N_h, *P_h;
    int height_m = 1024; // height of m
    int k = 1024;        // width of m; height of n.
    int width_n = 512;   // width of n

    size_t bytes_M = sizeof(float) * height_m * k;
    size_t bytes_N = sizeof(float) * k * width_n;
    size_t bytes_P = sizeof(float) * height_m * width_n;

    M_h = (float *)malloc(bytes_M);
    N_h = (float *)malloc(bytes_N);
    P_h = (float *)malloc(bytes_P);

    init_mat(M_h, height_m * k);
    init_mat(N_h, k * width_n);

    hipMalloc(&M_d, bytes_M);
    hipMalloc(&N_d, bytes_N);
    hipMalloc(&P_d, bytes_P);

    hipMemcpy(M_d, M_h, bytes_M, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, bytes_N, hipMemcpyHostToDevice);

    int THREADS = 16;
    int GRID_SIZE_X = (width_n + THREADS - 1) / THREADS;
    int GRID_SIZE_Y = (height_m + THREADS - 1) / THREADS;

    dim3 BLOCK_SIZE(THREADS, THREADS);
    dim3 GRID_SIZE(GRID_SIZE_X, GRID_SIZE_Y);

    matMultKernel<<<GRID_SIZE, BLOCK_SIZE>>>(M_d, N_d, P_d, height_m, k, width_n);

    hipMemcpy(P_h, P_d, bytes_P, hipMemcpyDeviceToHost);

    check_results(M_h, N_h, P_h, height_m, k, width_n);

    free(M_h);
    free(N_h);
    free(P_h);
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);

    printf("COMPLETED SUCCESSFULLY\n");
    return 0;
}