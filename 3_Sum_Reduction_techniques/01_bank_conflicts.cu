#include <iostream>
#include <cassert>
#include <cmath>
#include <hip/hip_runtime.h>

#define SIZE 256
#define SHMEM_SIZE SIZE * sizeof(int)

__global__ void sum_reduction(int *vector, int *vector_result)
{
    // shared memory block for partial sums
    __shared__ int partial_sum[SHMEM_SIZE];

    // global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // get the global element from vector (vector[tid]) to the corresponding local index of the SHMEM block.
    partial_sum[threadIdx.x] = vector[tid];
    // wait for all the threads to fetch and store the data in the SHMEM block.
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2)
    {
        int index = 2 * i * threadIdx.x;

        if (index < blockDim.x)
        {
            partial_sum[index] += partial_sum[index + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        vector_result[blockIdx.x] = partial_sum[0];
    }
}

void init_vector(int *vec, const int n)
{
    for (int i = 0; i < n; i++)
    {
        vec[i] = 1;
    }
}

int main()
{
    int N = 1 << 15;
    int bytes = N * sizeof(int);
    // host vetor and host vector result.
    int *h_v, *h_v_r;
    int *d_v, *d_v_r;

    h_v = (int *)malloc(bytes);
    h_v_r = (int *)malloc(bytes);

    hipMalloc(&d_v, bytes);
    hipMalloc(&d_v_r, bytes);

    init_vector(h_v, N);

    hipMemcpy(d_v, h_v, bytes, hipMemcpyHostToDevice);

    // thread block size <=> number of threads
    int threads_per_block = SIZE;

    // Grid size <=> number of threadblocks.
    int num_blocks = (int)ceil(N / threads_per_block);

    // Call kernel.
    sum_reduction<<<num_blocks, threads_per_block>>>(d_v, d_v_r);

    // reduce to scalar.
    sum_reduction<<<1, threads_per_block>>>(d_v_r, d_v_r);

    hipMemcpy(h_v_r, d_v_r, bytes, hipMemcpyDeviceToHost);

    printf("Accumulated result: %d\n", h_v_r[0]);

    assert(h_v_r[0] == 1 << 15);
    printf("COMPILATION SUCCEEDED\n");

    return 0;
}