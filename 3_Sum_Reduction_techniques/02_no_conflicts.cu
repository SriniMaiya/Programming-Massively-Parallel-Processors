#include <iostream>
#include <cassert>
#include <cmath>
#include <hip/hip_runtime.h>

#define SIZE 256
#define SHMEM_SIZE SIZE * sizeof(int)

__global__ void sum_reduction(int *vector, int *vector_result)
{
    // shared memory block for partial sums
    __shared__ int partial_sum[SHMEM_SIZE];

    // global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // get the global element from vector (vector[tid]) to the corresponding local index of the SHMEM block.
    partial_sum[threadIdx.x] = vector[tid];
    // wait for all the threads to fetch and store the data in the SHMEM block.
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i = i >> 1)
    {
        if (threadIdx.x < i)
        {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        vector_result[blockIdx.x] = partial_sum[0];
    }
}

void init_vector(int *vec, const int n)
{
    for (int i = 0; i < n; i++)
    {
        vec[i] = 1;
    }
}

int main()
{
    int N = 1 << 16;
    int bytes = N * sizeof(int);
    // host vetor and host vector result.
    int *h_v, *h_v_r;
    int *d_v, *d_v_r;

    h_v = (int *)malloc(bytes);
    h_v_r = (int *)malloc(bytes);

    hipMalloc(&d_v, bytes);
    hipMalloc(&d_v_r, bytes);

    init_vector(h_v, N);

    hipMemcpy(d_v, h_v, bytes, hipMemcpyHostToDevice);

    // thread block size <=> number of threads
    int threads_per_block = SIZE;

    // Grid size <=> number of threadblocks.
    int num_blocks = (int)ceil((float)N / threads_per_block);

    // Call kernel.
    sum_reduction<<<num_blocks, threads_per_block>>>(d_v, d_v_r);

    // reduce to scalar.
    sum_reduction<<<1, threads_per_block>>>(d_v_r, d_v_r);

    hipMemcpy(h_v_r, d_v_r, bytes, hipMemcpyDeviceToHost);

    printf("Accumulated result: %d\n", h_v_r[0]);

    assert(h_v_r[0] == 1 << 16);
    printf("COMPILATION SUCCEEDED\n");

    return 0;
}